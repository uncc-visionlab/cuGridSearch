/*
 * Copyright (C) 2022 Andrew R. Willis
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

/* system header */
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <vector>

/* MEX header */
#include <mex.h> 
#include "matrix.h"

/* nVIDIA CUDA header */
#include <hip/hip_runtime.h> 

/* fixing error : identifier "IUnknown" is undefined" */
#ifdef _WIN32
#define WIN32_LEAN_AND_MEAN
#endif

#if defined(_WIN32) || defined(_WIN64)
#include <windows.h>
#endif

//#define CUDAFUNCTION
#define CUDAFUNCTION __host__ __device__

/* includes CUDA kernel */
#include "cudaGridSearchKernels.cuh"

/* MEX entry function */
void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[])
 {
    double *A, *B, *C;
    mwSignedIndex Am, An, Bm, Bn;

    /* argument check */
    if (nrhs != 2) {
        mexErrMsgIdAndTxt("MATLAB:cudaAdd:inputmismatch",
                "Input arguments must be 2!");
    }
    if (nlhs != 1) {
        mexErrMsgIdAndTxt("MATLAB:cudaAdd:outputmismatch",
                "Output arguments must be 1!");
    }

    A = mxGetPr(prhs[0]);
    B = mxGetPr(prhs[1]);
    mexPrintf_800("%f\n",A[0]);
    mexPrintf_800("%f\n",B[1]);
    /* matrix size */
    Am = (mwSignedIndex) mxGetM(prhs[0]);
    An = (mwSignedIndex) mxGetN(prhs[0]);
    Bm = (mwSignedIndex) mxGetM(prhs[1]);
    Bn = (mwSignedIndex) mxGetN(prhs[1]);
    if (Am != Bm || An != Bn) {
        mexErrMsgIdAndTxt("MATLAB:cudaAdd:sizemismatch",
                "Input matrices must have the same size!");
    }

    /* allocate output */
    plhs[0] = mxCreateDoubleMatrix(Am, An, mxREAL);
    C = mxGetPr(plhs[0]);

    /* set GPU grid & block configuration */
    hipDeviceProp_t deviceProp;
    memset(&deviceProp, 0, sizeof (deviceProp));
    if (hipSuccess != hipGetDeviceProperties(&deviceProp, 0)) {
        mexPrintf_800("\n%s", hipGetErrorString(hipGetLastError()));
        return;
    }

    dim3 dimGridImg(8, 1, 1);
    dim3 dimBlockImg(1, 64, 1);

    /* allocate device memory for matrices */
    double *d_A = NULL;
    hipMalloc((void**) &d_A, Am * An * sizeof (double));
    hipMemcpy(d_A, A, Am * An * sizeof (double), hipMemcpyHostToDevice);
    double *d_B = NULL;
    hipMalloc((void**) &d_B, Bm * Bn * sizeof (double));
    hipMemcpy(d_B, B, Bm * Bn * sizeof (double), hipMemcpyHostToDevice);
    double *d_C = NULL;
    hipMalloc((void**) &d_C, Am * An * sizeof (double));

    /* call GPU kernel for addition */
    gpuAddKernel << < dimGridImg, dimBlockImg >>>(d_A, d_B, d_C, Am, An);
    hipDeviceSynchronize();

    /* copy result from device */
    hipMemcpy(C, d_C, Am * An * sizeof (double), hipMemcpyDeviceToHost);

    /* free GPU memory */
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

}
