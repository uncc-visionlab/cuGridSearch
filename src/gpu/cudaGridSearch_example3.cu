#include "hip/hip_runtime.h"
//#define CUDAFUNCTION
#define CUDAFUNCTION __host__ __device__

#include <cmath>
//#include <cstdlib>
#include <iostream>

#include "cudaImage.cuh"
#include "cudaGridSearch.cuh"
#include "cudaErrorFunctions.cuh"

#define grid_dimension 2        // the dimension of the grid, e.g., 1 => 1D grid, 2 => 2D grid, 3=> 3D grid, etc.
typedef float grid_precision;   // the type of values in the grid, e.g., float, double, int, etc.
typedef float func_precision;   // the type of values taken by the error function, e.g., float, double, int, etc.
typedef double pixel_precision; // the type of values in the image, e.g., float, double, int, etc.

typedef func_byvalue_t<func_precision, grid_precision, grid_dimension, CudaImage<pixel_precision>, CudaImage<pixel_precision> > image_err_func_byvalue;

// create device function pointer for by-value kernel function here
__device__ image_err_func_byvalue dev_func_byvalue_ptr = averageAbsoluteDifference<func_precision, grid_precision, grid_dimension, pixel_precision>;
//__device__ image_err_func_byvalue dev_func_byvalue_ptr = sumOfAbsoluteDifferences<func_precision, grid_precision, grid_dimension, pixel_precision>;

// test grid search
// classes typically store images in column major format so the images
// stored are the transpose of that shown in initialization below
pixel_precision imageA_data[6 * 6] = {0, 0, 0, 0, 0, 0,
                                      0, 0, 0, 0, 0, 0,
                                      0, 0, 1, 1, 0, 0,
                                      0, 0, 1, 1, 0, 0,
                                      0, 0, 0, 0, 0, 0,
                                      0, 0, 0, 0, 0, 0};

pixel_precision imageB_data[6 * 6] = {0, 0, 0, 0, 0, 0,
                                      0, 0, 0, 0, 0, 0,
                                      0, 0, 0, 0, 0, 0,
                                      0, 0, 0, 0, 0, 0,
                                      0, 0, 0, 0, 1, 1,
                                      0, 0, 0, 0, 1, 1};

pixel_precision imageC_data[6 * 6] = {1, 1, 0, 0, 0, 0,
                                      1, 1, 0, 0, 0, 0,
                                      0, 0, 0, 0, 0, 0,
                                      0, 0, 0, 0, 0, 0,
                                      0, 0, 0, 0, 0, 0,
                                      0, 0, 0, 0, 0, 0
};

int main(int argc, char **argv) {
    image_err_func_byvalue host_func_byval_ptr;

    int cuda_device = 0;
    hipDeviceProp_t deviceProp;
    cuda_device = findCudaDevice(0, nullptr);
    checkCudaErrors(hipGetDevice(&cuda_device));
    checkCudaErrors(hipGetDeviceProperties(&deviceProp, cuda_device));

    CudaImage<pixel_precision> m1(6, 6);
    CudaImage<pixel_precision> m2(6, 6);

    ck(hipMalloc(&m1._data, m1.bytesSize()));
    ck(hipMalloc(&m2._data, m2.bytesSize()));

    // Test here

    m1.setValuesFromVector(std::vector<pixel_precision>(imageA_data, imageA_data + 6 * 6));
    m2.setValuesFromVector(std::vector<pixel_precision>(imageA_data, imageA_data + 6 * 6));
//    m1.fill(5);
//    m2.fill(10);

    m1.display("m1");
    m2.display("m2");

    //m1 *= m2;
    //m1.display("m1 * m2");

    std::vector<grid_precision> start_point = {(grid_precision) -m2._width / 2, (grid_precision) -m2._height / 2};
    std::vector<grid_precision> end_point = {(grid_precision) std::abs(m1._width - (m2._width / 2)),
                                  (grid_precision) std::abs(m1._height - (m2._height / 2))};
    std::vector<grid_precision> resolution = {(grid_precision) 0.5f, (grid_precision) 0.5f};

    CudaGrid<grid_precision> translation_xy_grid(grid_dimension);
    ck(hipMalloc(&translation_xy_grid.data(), translation_xy_grid.bytesSize()));

    translation_xy_grid.setStartPoint(start_point);
    translation_xy_grid.setEndPoint(end_point);
    translation_xy_grid.setResolution(resolution);
    translation_xy_grid.display("translation_xy_grid");

    // first template argument is the error function return type
    // second template argument is the grid point value type
    CudaGridSearcher<func_precision, grid_precision> translation_xy_gridsearcher(translation_xy_grid);

    // Copy device function pointer for the function having by-value parameters to host side
    hipMemcpyFromSymbol(&host_func_byval_ptr, HIP_SYMBOL(dev_func_byvalue_ptr),
                         sizeof(image_err_func_byvalue));

    //translation_xy_gridsearcher.search(host_func_byval_ptr, m1, m2);
    translation_xy_gridsearcher.search_by_value(host_func_byval_ptr, m1, m2);

    // Clean memory
    ck(hipFree(m1._data));
    ck(hipFree(m2._data));
    ck(hipFree(translation_xy_grid.data()));

    return EXIT_SUCCESS;
}