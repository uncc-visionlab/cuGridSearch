#include "hip/hip_runtime.h"
//#define CUDAFUNCTION
#define CUDAFUNCTION __host__ __device__

#include <cmath>
//#include <cstdlib>
#include <iostream>

#include "cudaImage.cuh"
#include "cudaGridSearch.cuh"
#include "cudaErrorFunctions.cuh"

#define grid_dimension 2        // the dimension of the grid, e.g., 1 => 1D grid, 2 => 2D grid, 3=> 3D grid, etc.
typedef int32_t grid_precision;   // the type of values in the grid, e.g., float, double, int, etc.
typedef float func_precision;   // the type of values taken by the error function, e.g., float, double, int, etc.
typedef double pixel_precision; // the type of values in the image, e.g., float, double, int, etc.

typedef func_byvalue_t<func_precision, grid_precision, grid_dimension, CudaImage<pixel_precision>, CudaImage<pixel_precision> > image_err_func_byvalue;

// create device function pointer for by-value kernel function here
__device__ image_err_func_byvalue dev_func_byvalue_ptr = averageAbsoluteDifference<func_precision, grid_precision, grid_dimension, pixel_precision>;

// test grid search
// classes typically store images in column major format so the images
// stored are the transpose of that shown in initialization below
uint8_t imageA_data[6 * 6] = {0, 0, 0, 0, 0, 0,
                              0, 0, 0, 0, 0, 0,
                              0, 0, 1, 1, 0, 0,
                              0, 0, 1, 1, 0, 0,
                              0, 0, 0, 0, 0, 0,
                              0, 0, 0, 0, 0, 0};
uint8_t imageB_data[6 * 6] = {0, 0, 0, 0, 0, 0,
                              0, 0, 0, 0, 0, 0,
                              0, 0, 0, 0, 0, 0,
                              0, 0, 0, 0, 0, 0,
                              0, 0, 0, 0, 1, 1,
                              0, 0, 0, 0, 1, 1};
uint8_t imageC_data[6 * 6] = {
        1, 1, 0, 0, 0, 0,
        1, 1, 0, 0, 0, 0,
        0, 0, 0, 0, 0, 0,
        0, 0, 0, 0, 0, 0,
        0, 0, 0, 0, 0, 0,
        0, 0, 0, 0, 0, 0
};

int main(int argc, char **argv) {
    image_err_func_byvalue host_func_byval_ptr;

    int cuda_device = 0;
    hipDeviceProp_t deviceProp;
    cuda_device = findCudaDevice(0, nullptr);
    checkCudaErrors(hipGetDevice(&cuda_device));
    checkCudaErrors(hipGetDeviceProperties(&deviceProp, cuda_device));

    CudaImage<pixel_precision> m1(6, 6);
    CudaImage<pixel_precision> m2(6, 6);

    ck(hipMalloc(&m1._data, m1.bytesSize()));
    ck(hipMalloc(&m2._data, m2.bytesSize()));

    // Test here

    //m1.setValuesFromVector({1, 1, 1, 2, 2, 2, 3, 3, 3});
    m1.fill(5);
    m2.fill(10);

    //m1.display("m1");
    //m2.display("m2");

    // Fails here
    //m1 *= m2;

    m1.display("m1 * m2");

    std::vector<Bounds<grid_precision>> bounds;
    bounds.push_back(Bounds<grid_precision>(-m2._width / 2, 1, std::abs(m1._width - 1 - (m2._width / 2))));
    bounds.push_back(Bounds<grid_precision>(-m2._height / 2, 1, std::abs(m1._height - 1 - (m2._height / 2))));

    //Grid<int32_t> translation_xy(2);
    Grid<grid_precision> translation_xy(grid_dimension, bounds);

    std::vector<float> search_resolution = {1.0f, 1.0f};
    translation_xy.setResolution(search_resolution);

    // first template argument is the error function return type
    // second template argument is the grid point value type
    CudaGridSearcher<func_precision, grid_precision> translation_xy_searcher(translation_xy);

    // Copy device function pointer for the function having by-value parameters to host side
    hipMemcpyFromSymbol(&host_func_byval_ptr, HIP_SYMBOL(dev_func_byvalue_ptr),
                         sizeof(image_err_func_byvalue));

    //translation_xy_searcher.search(host_func_byval_ptr, m1, m2);
    translation_xy_searcher.search_by_value(host_func_byval_ptr, m1, m2);

    // Clean memory
    ck(hipFree(m1._data));
    ck(hipFree(m2._data));

    return EXIT_SUCCESS;
}