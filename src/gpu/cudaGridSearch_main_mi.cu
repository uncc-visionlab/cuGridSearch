#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2022 Andrew R. Willis
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

/* system header */
#include <cmath>
#include <iostream>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <vector>

/* nVIDIA CUDA header */
#include <hip/hip_runtime.h>

#include <cxxopts.hpp>

//#define CUDAFUNCTION
#define CUDAFUNCTION __host__ __device__

#include "helper_functions.h"
#include "hip/hip_runtime_api.h"

#include "cudaTensor.cuh"

#include "cudaGridSearch.cuh"
#include "cudaErrorFunctions.cuh"
#include "cudaErrorFunctionsStreams.cuh"
#include "cudaErrorFunction_miStreams.cuh"
#include "cudaErrorFunction_mi.cuh"

#define STB_IMAGE_IMPLEMENTATION
#define STBI_FAILURE_USERMSG
#define STBI_NO_FAILURE_STRINGS

#include "stb_image.h"

#define STB_IMAGE_WRITE_IMPLEMENTATION

#include "stb_image_write.h"

#define PI 3.14159265

template<typename pixType, uint8_t D, uint8_t CHANNELS>
__global__ void transformImage(nv_ext::Vec<float, D> H,
                               CudaImage<pixType, CHANNELS> img_in,
                               CudaImage<pixType, CHANNELS> img_out) {
    int colsm = img_in.width();
    int rowsm = img_in.height();
    // In case H < 4 //Similarity transform
    float cpH[4] = {0.0, 0.0, 0.0, 1.0};
    for (int i = 0; i < D; i++) {
        cpH[i] = H[i];
    }
    // Transform the image
    float ct = cos(H[2]);
    float st = sin(H[2]);
    for (int x = 0; x < colsm; x++) {
        for (int y = 0; y < rowsm; y++) {
            for (int c = 0; c < CHANNELS; c++) {
                float new_x = (cpH[3] * (x - colsm / 2) * ct - (y - rowsm / 2) * st + cpH[0] + cpH[3] * colsm / 2);
                float new_y = ((x - colsm / 2) * st + cpH[3] * (y - rowsm / 2) * ct + cpH[1] + cpH[3] * rowsm / 2);
                if (img_out.inImage(new_y, new_x)) {
                    img_out.template at<float>(new_y, new_x) = img_in.template at<float>(y, x);
                }
            }
        }
    }
}

#define grid_dimension 4        // the dimension of the grid, e.g., 1 => 1D grid, 2 => 2D grid, 3=> 3D grid, etc.
#define CHANNELS 1
#define DEPTH 1
typedef float grid_precision;   // the type of values in the grid, e.g., float, double, int, etc.
typedef float func_precision;   // the type of values taken by the error function, e.g., float, double, int, etc.
typedef uint8_t pixel_precision; // the type of values in the image, e.g., float, double, int, etc.

// typedef func_byvalue_t<func_precision, grid_precision, grid_dimension, CudaImage<pixel_precision>, CudaImage<pixel_precision> > image_err_func_byvalue;

// create device function pointer for by-value kernel function here
// __device__ image_err_func_byvalue dev_func_byvalue_ptr = averageAbsoluteDifference<func_precision, grid_precision, grid_dimension, pixel_precision>;
//__device__ image_err_func_byvalue dev_func_byvalue_ptr = sumOfAbsoluteDifferences<func_precision, grid_precision, grid_dimension, pixel_precision>;

// grid_mi
// typedef func_byvalue_t<func_precision, grid_precision, grid_dimension,
//         CudaImage<pixel_precision, CHANNELS>, CudaImage<pixel_precision, CHANNELS> > image_err_func_byvalue;

// calcMIstream
typedef func_byvalue_t<func_precision, grid_precision, grid_dimension,
        CudaImage<pixel_precision, CHANNELS>, CudaImage<pixel_precision, CHANNELS>, CudaImage<float, 1>, CudaImage<float, 1>, CudaImage<float, 1> > image_err_func_byvalue;

// __device__ image_err_func_byvalue dev_func_byvalue_ptr = grid_miStream<func_precision, grid_precision,
//         grid_dimension, CHANNELS, pixel_precision>;
__device__ image_err_func_byvalue dev_func_byvalue_ptr = calcMIstream<func_precision, grid_precision,
        grid_dimension, CHANNELS, pixel_precision>;
//__device__ image_err_func_byvalue dev_func_byvalue_ptr = calcMI<func_precision, grid_precision,
//        grid_dimension, CHANNELS, pixel_precision>;

#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

void cxxopts_integration(cxxopts::Options &options) {

    options.add_options()
            ("i_ref", "Reference Image (image in the reference coordinate frame)", cxxopts::value<std::string>())
            ("i_mov", "Moved Image (image in the measured coordinate frame)", cxxopts::value<std::string>())
            ("d,debug", "Enable debugging", cxxopts::value<bool>()->default_value("false"))
            ("r,dynrange", "Dynamic Range (dB) <70 dB>", cxxopts::value<float>()->default_value("70"))
            ("o,output", "Output file <output_image.png>",
             cxxopts::value<std::string>()->default_value("output_image.png"))
            ("h,help", "Print usage");
}

void printMatrix(double **matrix, int ROWS, int COLUMNS) {

    for (int r = 0; r < ROWS; r++) {
        for (int c = 0; c < COLUMNS; c++) {
            std::cout << matrix[r][c] << " ";
        }
        std::cout << std::endl;
    }
}

/*
 * Case Insensitive Implementation of endsWith()
 * It checks if the string 'mainStr' ends with given string 'toMatch'
 */
bool endsWithCaseInsensitive(std::string mainStr, std::string toMatch) {
    auto it = toMatch.begin();
    return mainStr.size() >= toMatch.size() &&
           std::all_of(std::next(mainStr.begin(), mainStr.size() - toMatch.size()), mainStr.end(),
                       [&it](const char &c) {
                           return ::tolower(c) == ::tolower(*(it++));
                       });
}

// test grid search
// classes typically store images in column major format so the images
// stored are the transpose of that shown in initialization below
pixel_precision imageA_data[6 * 6] = {0, 0, 0, 0, 0, 0,
                                      0, 0, 0, 0, 0, 0,
                                      0, 0, 1, 1, 0, 0,
                                      0, 0, 1, 1, 0, 0,
                                      0, 0, 0, 0, 0, 0,
                                      0, 0, 0, 0, 0, 0};

pixel_precision imageB_data[6 * 6] = {0, 0, 0, 0, 0, 0,
                                      0, 0, 0, 0, 0, 0,
                                      0, 0, 0, 0, 0, 0,
                                      0, 0, 0, 0, 0, 0,
                                      0, 0, 0, 0, 1, 1,
                                      0, 0, 0, 0, 1, 1};

pixel_precision imageC_data[6 * 6] = {1, 1, 0, 0, 0, 0,
                                      1, 1, 0, 0, 0, 0,
                                      0, 0, 0, 0, 0, 0,
                                      0, 0, 0, 0, 0, 0,
                                      0, 0, 0, 0, 0, 0,
                                      0, 0, 0, 0, 0, 0
};

int main(int argc, char **argv) {

    // Argument parsing
    cxxopts::Options options("cuda_gridsearch", "UNC Charlotte Machine Vision Lab CUDA-accelerated grid search code.");
    cxxopts_integration(options);
    auto result = options.parse(argc, argv);
    std::string img_fixed_filename, img_moved_filename, img_out_filename;
    if (result.count("i_ref")) {
        img_fixed_filename = result["i_ref"].as<std::string>();
    } else {
        std::cerr << "No input reference image filename was provided. Exiting.." << std::endl;
        return EXIT_FAILURE;
    }
    if (result.count("i_mov")) {
        img_moved_filename = result["i_mov"].as<std::string>();
    } else {
        std::cerr << "No input moving image filename was provided. Exiting.." << std::endl;
        return EXIT_FAILURE;
    }
    img_out_filename = result["output"].as<std::string>();
    std::cerr << "Output image filename is " << img_out_filename << "." << std::endl;
    if (result.count("help")) {
        std::cout << options.help() << std::endl;
        return EXIT_SUCCESS;
    }

    /* set GPU grid & block configuration */
    image_err_func_byvalue host_func_byval_ptr;
    int cuda_device = 0;
    hipDeviceProp_t deviceProp;

    cuda_device = findCudaDevice(0, nullptr);
    checkCudaErrors(hipGetDevice(&cuda_device));
    checkCudaErrors(hipGetDeviceProperties(&deviceProp, cuda_device));

    // Load input images from disk
    int xf, yf, nf;
    uint8_t *dataf = stbi_load(img_fixed_filename.c_str(), &xf, &yf, &nf, CHANNELS);
    if (dataf == NULL) {
        std::cerr << "Reference image " + img_fixed_filename + " failed to load!" << std::endl;
        return EXIT_FAILURE;
    }
    int xm, ym, nm;
    uint8_t *datam = stbi_load(img_moved_filename.c_str(), &xm, &ym, &nm, CHANNELS);
    if (datam == NULL) {
        std::cerr << "Moving image " + img_moved_filename + " failed to load!" << std::endl;
        return EXIT_FAILURE;
    }

    // number of components must be equal on construction
    printf("%d %d\n", nf, nm);
    //assert(nf == CHANNELS && nm == CHANNELS);

    CudaImage<uint8_t, CHANNELS> image_fix(yf, xf);
    CudaImage<uint8_t, CHANNELS> image_mov(ym, xm);

    int binN = 64;
    float h_px[binN] = {0};
    CudaImage<float, 1> d_px(binN, 1);
    CudaImage<float, 1> d_py(binN, 1);
    CudaImage<float, 1> d_pxy(binN, binN);

    for (int i = 0; i < yf * xf; i++) {
        int temp = dataf[i] / (256 / binN);
        h_px[temp] += (1.0f / (yf * xf));
    }

    checkCudaErrors(hipMalloc(&d_px.data(), d_px.bytesSize()));
    checkCudaErrors(hipMalloc(&d_py.data(), d_py.bytesSize()));
    checkCudaErrors(hipMalloc(&d_pxy.data(), d_pxy.bytesSize()));

    checkCudaErrors(hipMemcpy(d_px.data(), h_px, d_px.bytesSize(), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemset(d_py.data(), 0, d_py.bytesSize()));
    checkCudaErrors(hipMemset(d_pxy.data(), 0, d_pxy.bytesSize()));

    checkCudaErrors(hipMalloc(&image_fix.data(), image_fix.bytesSize()));
    checkCudaErrors(hipMemcpy(image_fix.data(), dataf, image_fix.bytesSize(), hipMemcpyHostToDevice));

    checkCudaErrors(hipMalloc(&image_mov.data(), image_mov.bytesSize()));
    checkCudaErrors(hipMemcpy(image_mov.data(), datam, image_mov.bytesSize(), hipMemcpyHostToDevice));

    stbi_image_free(dataf);
    stbi_image_free(datam);

    checkCudaErrors(hipDeviceSetLimit(hipLimitMallocHeapSize, (size_t) (1 << 30)));

    // Example
    // Fixed 581x593
    // Moving 100x100
    // Total samples 276,342,848
    // std::vector<grid_precision> start_point = {(grid_precision) 100, (grid_precision) 200, (grid_precision) 0, (grid_precision) 1};
    // std::vector<grid_precision> end_point =   {(grid_precision) 300, (grid_precision) 400, (grid_precision) (2*PI)-(PI/180), (grid_precision) 5};
    // std::vector<grid_precision> num_samples =  {(grid_precision) 201, (grid_precision) 201, (grid_precision) 360, (grid_precision) 9};

    // // Example MI
    std::vector<grid_precision> start_point = {(grid_precision) -xm / 2, (grid_precision) -ym / 2, (grid_precision) 0,
                                               (grid_precision) 1};
    std::vector<grid_precision> end_point = {(grid_precision) xf - xm / 2, (grid_precision) yf - ym / 2,
                                             (grid_precision) 0, (grid_precision) 1};
    std::vector<grid_precision> num_samples = {(grid_precision) (xf + 1) / 4, (grid_precision) (yf + 1) / 4,
                                               (grid_precision) 1,
                                               (grid_precision) 1};

    CudaGrid<grid_precision, grid_dimension> affineTransform_grid;
    checkCudaErrors(hipMalloc(&affineTransform_grid.data(), affineTransform_grid.bytesSize()));

    for (int iii = 0; iii < DEPTH; iii++) {
        affineTransform_grid.setStartPoint(start_point);
        affineTransform_grid.setEndPoint(end_point);
        affineTransform_grid.setNumSamples(num_samples);
        affineTransform_grid.display("affineTransform_grid");

        grid_precision axis_sample_counts[grid_dimension];
        affineTransform_grid.getAxisSampleCounts(axis_sample_counts);

        CudaTensor<func_precision, grid_dimension> func_values(axis_sample_counts);
        checkCudaErrors(hipMalloc(&func_values.data(), func_values.bytesSize()));
        //func_values.fill(0);

        // first template argument is the error function return type
        // second template argument is the grid point value type
        CudaGridSearcher<func_precision, grid_precision, grid_dimension> affineTransform_gridsearcher(
                affineTransform_grid,
                func_values);

        // Mutual Information
        // Copy device function pointer for the function having by-value parameters to host side
        hipMemcpyFromSymbol(&host_func_byval_ptr, HIP_SYMBOL(dev_func_byvalue_ptr),
                             sizeof(image_err_func_byvalue));

        //affineTransform_gridsearcher.search(host_func_byval_ptr, m1, m2);
        // affineTransform_gridsearcher.search_by_value(host_func_byval_ptr, m1, m2);
        affineTransform_gridsearcher.search_by_value_stream(host_func_byval_ptr, 10000, image_mov.height(), image_mov,
                                                            image_fix, d_px, d_py, d_pxy);
        // affineTransform_gridsearcher.search_by_value_stream(host_func_byval_ptr, 10000, 1, image_mov, image_fix);
//        affineTransform_gridsearcher.search_by_value(host_func_byval_ptr, image_mov, image_fix, d_px, d_py, d_pxy);

        //    func_values.display();

        func_precision min_value;
        int32_t min_value_index1d;
        func_values.find_extrema(min_value, min_value_index1d);

        grid_precision min_grid_point[grid_dimension];
        affineTransform_grid.getGridPoint(min_grid_point, min_value_index1d);
        std::cout << "Minimum found at point p = { ";
        for (int d = 0; d < grid_dimension; d++) {
            std::cout << min_grid_point[d] << ((d < grid_dimension - 1) ? ", " : " ");
            if (num_samples[d] / 2 > 2) {
                start_point[d] = min_grid_point[d] - (end_point[d] - start_point[d]) / 4;
                end_point[d] = min_grid_point[d] + (end_point[d] - start_point[d]) / 4;
                num_samples[d] = ceil(num_samples[d] / 2);
            } else {
                start_point[d] = min_grid_point[d];
                end_point[d] = min_grid_point[d];
                num_samples[d] = 1;
            }
        }
        std::cout << "}" << std::endl;

        checkCudaErrors(hipFree(func_values.data()));
    }

    // Write an output image to disk
    CudaImage<uint8_t, CHANNELS> image_out(ym, xm);
    checkCudaErrors(hipMalloc(&image_out.data(), image_out.bytesSize()));
    checkCudaErrors(hipMemset(image_out.data(), 0, image_out.bytesSize()));
    nv_ext::Vec<float, 4> H;
    H[0] = H[1] = 50; H[2] = 1.0; H[3] = 1.0;
    transformImage<uint8_t, 4, CHANNELS><<<1, 1>>>(H, image_mov, image_out);

    pixel_precision *hostValues;
    checkCudaErrors(hipHostMalloc(&hostValues, image_out.bytesSize()));
    checkCudaErrors(hipMemcpy(hostValues, image_out.data(), image_out.bytesSize(), hipMemcpyDeviceToHost));
    if (endsWithCaseInsensitive(img_out_filename, ".png")) {
        stbi_write_png(img_out_filename.c_str(), image_out.width(), image_out.height(), CHANNELS, hostValues,
                       image_fix.width() * sizeof(pixel_precision) * CHANNELS);
        // You have to use 3 comp for complete jpg file. If not, the image will be grayscale or nothing.
    } else if (endsWithCaseInsensitive(img_out_filename, ".jpg")) {
        stbi_write_jpg(img_out_filename.c_str(), image_out.width(), image_out.height(), CHANNELS, hostValues, 95);
    } else {
        std::cout << "Filename suffix has image format not recognized." << std::endl;
    }
    hipHostFree(hostValues);

    // Clean memory
    checkCudaErrors(hipFree(image_fix.data()));
    checkCudaErrors(hipFree(image_mov.data()));
    checkCudaErrors(hipFree(affineTransform_grid.data()));
    checkCudaErrors(hipFree(d_px.data()));
    checkCudaErrors(hipFree(d_py.data()));
    checkCudaErrors(hipFree(d_pxy.data()));
    return EXIT_SUCCESS;
}
