#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2022 Andrew R. Willis
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

/* system header */
#include <cmath>
#include <iostream>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <vector>

/* nVIDIA CUDA header */
#include <hip/hip_runtime.h>

#include <cxxopts.hpp>

//#define CUDAFUNCTION
#define CUDAFUNCTION __host__ __device__

#include "helper_functions.h"
#include "hip/hip_runtime_api.h"

#include "cudaTensor.cuh"

#include "cudaGridSearch.cuh"
#include "cudaErrorFunctions.cuh"
#include "cudaErrorFunctionsStreams.cuh"
#include "cudaErrorFunction_miStreams.cuh"
#include "cudaErrorFunction_mi.cuh"
#include "cudaImageFunctions.cuh"

#define STB_IMAGE_IMPLEMENTATION
#define STBI_FAILURE_USERMSG
#define STBI_NO_FAILURE_STRINGS

#include "stb_image.h"

#define PI 3.14159265
#define grid_dimension 4        // the dimension of the grid, e.g., 1 => 1D grid, 2 => 2D grid, 3=> 3D grid, etc.
#define CHANNELS 1
#define DEPTH 1
typedef float grid_precision;   // the type of values in the grid, e.g., float, double, int, etc.
typedef float func_precision;   // the type of values taken by the error function, e.g., float, double, int, etc.
typedef uint8_t pixel_precision; // the type of values in the image, e.g., float, double, int, etc.

// typedef func_byvalue_t<func_precision, grid_precision, grid_dimension, CudaImage<pixel_precision>, CudaImage<pixel_precision> > image_err_func_byvalue;

// create device function pointer for by-value kernel function here
// __device__ image_err_func_byvalue dev_func_byvalue_ptr = averageAbsoluteDifference<func_precision, grid_precision, grid_dimension, pixel_precision>;
//__device__ image_err_func_byvalue dev_func_byvalue_ptr = sumOfAbsoluteDifferences<func_precision, grid_precision, grid_dimension, pixel_precision>;

// grid_mi
// typedef func_byvalue_t<func_precision, grid_precision, grid_dimension,
//         CudaImage<pixel_precision, CHANNELS>, CudaImage<pixel_precision, CHANNELS> > image_err_func_byvalue;

// calcMIstream
typedef func_byvalue_t<func_precision, grid_precision, grid_dimension,
        CudaImage<pixel_precision, CHANNELS>, CudaImage<pixel_precision, CHANNELS>, CudaImage<float, 1>, CudaImage<float, 1>, CudaImage<float, 1> > image_err_func_byvalue;

// __device__ image_err_func_byvalue dev_func_byvalue_ptr = grid_miStream<func_precision, grid_precision,
//         grid_dimension, CHANNELS, pixel_precision>;
__device__ image_err_func_byvalue dev_func_byvalue_ptr = calcMIstream<func_precision, grid_precision,
        grid_dimension, CHANNELS, pixel_precision>;
//__device__ image_err_func_byvalue dev_func_byvalue_ptr = calcMI<func_precision, grid_precision,
//        grid_dimension, CHANNELS, pixel_precision>;

#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

void cxxopts_integration(cxxopts::Options &options) {

    options.add_options()
            ("i_ref", "Reference Image (image in the reference coordinate frame)", cxxopts::value<std::string>())
            ("i_mov", "Moved Image (image in the measured coordinate frame)", cxxopts::value<std::string>())
            ("d,debug", "Enable debugging", cxxopts::value<bool>()->default_value("false"))
            ("o,output", "Output file <output_image.png>",
             cxxopts::value<std::string>()->default_value("output_image.png"))
            ("f,fusedoutput", "Fused output file <output_image_fused.png>",
             cxxopts::value<std::string>()->default_value("output_image_fused.png"))
            ("h,help", "Print usage");
}

void printMatrix(double **matrix, int ROWS, int COLUMNS) {

    for (int r = 0; r < ROWS; r++) {
        for (int c = 0; c < COLUMNS; c++) {
            std::cout << matrix[r][c] << " ";
        }
        std::cout << std::endl;
    }
}

// test grid search
// classes typically store images in column major format so the images
// stored are the transpose of that shown in initialization below
pixel_precision imageA_data[6 * 6] = {0, 0, 0, 0, 0, 0,
                                      0, 0, 0, 0, 0, 0,
                                      0, 0, 1, 1, 0, 0,
                                      0, 0, 1, 1, 0, 0,
                                      0, 0, 0, 0, 0, 0,
                                      0, 0, 0, 0, 0, 0};

pixel_precision imageB_data[6 * 6] = {0, 0, 0, 0, 0, 0,
                                      0, 0, 0, 0, 0, 0,
                                      0, 0, 0, 0, 0, 0,
                                      0, 0, 0, 0, 0, 0,
                                      0, 0, 0, 0, 1, 1,
                                      0, 0, 0, 0, 1, 1};

pixel_precision imageC_data[6 * 6] = {1, 1, 0, 0, 0, 0,
                                      1, 1, 0, 0, 0, 0,
                                      0, 0, 0, 0, 0, 0,
                                      0, 0, 0, 0, 0, 0,
                                      0, 0, 0, 0, 0, 0,
                                      0, 0, 0, 0, 0, 0
};

int main(int argc, char **argv) {

    // Argument parsing
    cxxopts::Options options("cuda_gridsearch", "UNC Charlotte Machine Vision Lab CUDA-accelerated grid search code.");
    cxxopts_integration(options);
    auto result = options.parse(argc, argv);
    std::string img_fixed_filename, img_moved_filename, img_out_filename, img_fused_filename;
    if (result.count("i_ref")) {
        img_fixed_filename = result["i_ref"].as<std::string>();
    } else {
        std::cerr << "No input reference image filename was provided. Exiting.." << std::endl;
        return EXIT_FAILURE;
    }
    if (result.count("i_mov")) {
        img_moved_filename = result["i_mov"].as<std::string>();
    } else {
        std::cerr << "No input moving image filename was provided. Exiting.." << std::endl;
        return EXIT_FAILURE;
    }
    img_out_filename = result["output"].as<std::string>();
    img_fused_filename = result["fusedoutput"].as<std::string>();
    std::cerr << "Output image filename is " << img_out_filename << "." << std::endl;
    std::cerr << "Fused Output image filename is " << img_fused_filename << "." << std::endl;
    if (result.count("help")) {
        std::cout << options.help() << std::endl;
        return EXIT_SUCCESS;
    }

    /* set GPU grid & block configuration */
    image_err_func_byvalue host_func_byval_ptr;
    int cuda_device = 0;
    hipDeviceProp_t deviceProp;

    cuda_device = findCudaDevice(0, nullptr);
    checkCudaErrors(hipGetDevice(&cuda_device));
    checkCudaErrors(hipGetDeviceProperties(&deviceProp, cuda_device));

    // Load input images from disk
    int xf, yf, nf;
    uint8_t *dataf = stbi_load(img_fixed_filename.c_str(), &xf, &yf, &nf, CHANNELS);
    if (dataf == NULL) {
        std::cerr << "Reference image " + img_fixed_filename + " failed to load!" << std::endl;
        return EXIT_FAILURE;
    }
    int xm, ym, nm;
    uint8_t *datam = stbi_load(img_moved_filename.c_str(), &xm, &ym, &nm, CHANNELS);
    if (datam == NULL) {
        std::cerr << "Moving image " + img_moved_filename + " failed to load!" << std::endl;
        return EXIT_FAILURE;
    }

    // number of components must be equal on construction
    printf("%d %d\n", nf, nm);
    //assert(nf == CHANNELS && nm == CHANNELS);

    CudaImage<uint8_t, CHANNELS> image_fix(yf, xf);
    CudaImage<uint8_t, CHANNELS> image_mov(ym, xm);

    int binN = 64;
    float h_px[binN] = {0};
    CudaImage<float, 1> d_px(binN, 1);
    CudaImage<float, 1> d_py(binN, 1);
    CudaImage<float, 1> d_pxy(binN, binN);

    for (int i = 0; i < yf * xf; i++) {
        int temp = dataf[i] / (256 / binN);
        h_px[temp] += (1.0f / (yf * xf));
    }

    checkCudaErrors(hipMalloc(&d_px.data(), d_px.bytesSize()));
    checkCudaErrors(hipMalloc(&d_py.data(), d_py.bytesSize()));
    checkCudaErrors(hipMalloc(&d_pxy.data(), d_pxy.bytesSize()));

    checkCudaErrors(hipMemcpy(d_px.data(), h_px, d_px.bytesSize(), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemset(d_py.data(), 0, d_py.bytesSize()));
    checkCudaErrors(hipMemset(d_pxy.data(), 0, d_pxy.bytesSize()));

    checkCudaErrors(hipMalloc(&image_fix.data(), image_fix.bytesSize()));
    checkCudaErrors(hipMemcpy(image_fix.data(), dataf, image_fix.bytesSize(), hipMemcpyHostToDevice));

    checkCudaErrors(hipMalloc(&image_mov.data(), image_mov.bytesSize()));
    checkCudaErrors(hipMemcpy(image_mov.data(), datam, image_mov.bytesSize(), hipMemcpyHostToDevice));

    stbi_image_free(dataf);
    stbi_image_free(datam);

    checkCudaErrors(hipDeviceSetLimit(hipLimitMallocHeapSize, (size_t) (1 << 30)));

    // Example
    // Fixed 581x593
    // Moving 100x100
    // Total samples 276,342,848
    // std::vector<grid_precision> start_point = {(grid_precision) 100, (grid_precision) 200, (grid_precision) 0, (grid_precision) 1};
    // std::vector<grid_precision> end_point =   {(grid_precision) 300, (grid_precision) 400, (grid_precision) (2*PI)-(PI/180), (grid_precision) 5};
    // std::vector<grid_precision> num_samples =  {(grid_precision) 201, (grid_precision) 201, (grid_precision) 360, (grid_precision) 9};

    // // Example MI
    std::vector<grid_precision> start_point = {(grid_precision) -xm / 2, (grid_precision) -ym / 2, (grid_precision) 0,
                                               (grid_precision) 1};
    std::vector<grid_precision> end_point = {(grid_precision) xf - xm / 2, (grid_precision) yf - ym / 2,
                                             (grid_precision) 0, (grid_precision) 1};
    std::vector<grid_precision> num_samples = {(grid_precision) (xf + 1) / 14, (grid_precision) (yf + 1) / 14,
                                               (grid_precision) 1,
                                               (grid_precision) 1};

    CudaGrid<grid_precision, grid_dimension> affineTransform_grid;
    checkCudaErrors(hipMalloc(&affineTransform_grid.data(), affineTransform_grid.bytesSize()));

    for (int iii = 0; iii < DEPTH; iii++) {
        affineTransform_grid.setStartPoint(start_point);
        affineTransform_grid.setEndPoint(end_point);
        affineTransform_grid.setNumSamples(num_samples);
        affineTransform_grid.display("affineTransform_grid");

        grid_precision axis_sample_counts[grid_dimension];
        affineTransform_grid.getAxisSampleCounts(axis_sample_counts);

        CudaTensor<func_precision, grid_dimension> func_values(axis_sample_counts);
        checkCudaErrors(hipMalloc(&func_values.data(), func_values.bytesSize()));
        //func_values.fill(0);

        // first template argument is the error function return type
        // second template argument is the grid point value type
        CudaGridSearcher<func_precision, grid_precision, grid_dimension> affineTransform_gridsearcher(
                affineTransform_grid,
                func_values);

        // Mutual Information
        // Copy device function pointer for the function having by-value parameters to host side
        hipMemcpyFromSymbol(&host_func_byval_ptr, HIP_SYMBOL(dev_func_byvalue_ptr),
                             sizeof(image_err_func_byvalue));

        //affineTransform_gridsearcher.search(host_func_byval_ptr, m1, m2);
        // affineTransform_gridsearcher.search_by_value(host_func_byval_ptr, m1, m2);
        affineTransform_gridsearcher.search_by_value_stream(host_func_byval_ptr, 10000, image_mov.height(), image_mov,
                                                            image_fix, d_px, d_py, d_pxy);
        // affineTransform_gridsearcher.search_by_value_stream(host_func_byval_ptr, 10000, 1, image_mov, image_fix);
//        affineTransform_gridsearcher.search_by_value(host_func_byval_ptr, image_mov, image_fix, d_px, d_py, d_pxy);

        //    func_values.display();

        func_precision min_value;
        int32_t min_value_index1d;
        func_values.find_extrema(min_value, min_value_index1d);

        grid_precision min_grid_point[grid_dimension];
        affineTransform_grid.getGridPoint(min_grid_point, min_value_index1d);
        std::cout << "Minimum found at point p = { ";
        for (int d = 0; d < grid_dimension; d++) {
            std::cout << min_grid_point[d] << ((d < grid_dimension - 1) ? ", " : " ");
            if (num_samples[d] / 2 > 2) {
                start_point[d] = min_grid_point[d] - (end_point[d] - start_point[d]) / 4;
                end_point[d] = min_grid_point[d] + (end_point[d] - start_point[d]) / 4;
                num_samples[d] = ceil(num_samples[d] / 2);
            } else {
                start_point[d] = min_grid_point[d];
                end_point[d] = min_grid_point[d];
                num_samples[d] = 1;
            }
        }
        std::cout << "}" << std::endl;

        checkCudaErrors(hipFree(func_values.data()));
    }

    //    linear interpolation in homography / affine matrix space
    //    https://math.stackexchange.com/questions/612006/decomposing-an-affine-transformation
    //    using the parameterization described by Stephane Laurent
    float theta = 5.0 * PI / 180.0; // range [0, 2*PI]
    float scaleX = 1.5;  // // range [1, 2]
    float scaleY = 1.5;  // // range [1, 2]
    float shearXY = 0.2; // range [-0.2, 0.2]
    float translateX = -10; // range [-image.width()/2, image.width()/2]
    float translateY = -30; // range [-image.height()/2, image.height()/2]
    float keystoneX = 0.0; // range [-0.1, 0.1]
    float keystoneY = 0.0; // range [-0.1, 0.1]
    // Transform does scale, shear, rotate, then translate and finally perspective project per the website
    // https://math.stackexchange.com/questions/612006/decomposing-an-affine-transformation
    //float initialH[] = {1.0, 0.0, 0.0, 0.0, 1.0, 0.0, 0.0, 0.0};
    float initialH[] = {scaleX * cos(theta), scaleY * shearXY * cos(theta) - scaleY * sin(theta), scaleX * translateX,
                        scaleX * sin(theta), scaleY * shearXY * sin(theta) + scaleY * cos(theta), scaleY * translateY,
                        keystoneX, keystoneY};
    nv_ext::Vec<float, 8> H(initialH);

    // Write an output image to disk
    writeTransformedImageToDisk<uint8_t, CHANNELS>(image_mov, H, img_out_filename);

    // Write aligned and fused output image to disk
    writeAlignedAndFusedImageToDisk<uint8_t, CHANNELS>(image_fix, image_mov, H, H, img_fused_filename);

    // Clean memory
    checkCudaErrors(hipFree(image_fix.data()));
    checkCudaErrors(hipFree(image_mov.data()));
    checkCudaErrors(hipFree(affineTransform_grid.data()));
    checkCudaErrors(hipFree(d_px.data()));
    checkCudaErrors(hipFree(d_py.data()));
    checkCudaErrors(hipFree(d_pxy.data()));
    return EXIT_SUCCESS;
}
