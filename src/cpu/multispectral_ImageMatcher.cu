#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2022 Andrew R. Willis
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

/* system header */


// #include <Eigen/Dense>

// #include <unsupported/Eigen/NonLinearOptimization>
// #include <unsupported/Eigen/NumericalDiff>

#include <cmath>
#include <iostream>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <vector>
#include <fstream>
#include <chrono>

/* nVIDIA CUDA header */
//#include <hip/hip_runtime.h>

#include "third_party/cxxopts.hpp"

//#define CUDAFUNCTION
//#define CUDAFUNCTION __host__ __device__
//
//#include "helper_functions.h"
//#include "hip/hip_runtime_api.h"
#include "cuGridSearch.cuh"
//#include "cudaTensor.cuh"
//
//#include "../gpu/cudaGridSearch.cuh"
//#include "../gpu/cudaErrorFunctions.cuh"
#include "../gpu/cudaErrorFunction_mi.cuh"
//#include "../gpu/cudaErrorFunctionsStreams.cuh"
//#include "../gpu/cudaErrorFunction_miStreams.cuh"
#include "../gpu/cudaImageFunctions.cuh"

#define STB_IMAGE_IMPLEMENTATION
#define STBI_FAILURE_USERMSG
#define STBI_NO_FAILURE_STRINGS

#include "stb_image.h"

#define STB_IMAGE_RESIZE_IMPLEMENTATION

#include "stb_image_resize.h"

#define PI 3.14159265

#define DEBUG true

#define grid_dimension 8        // the dimension of the grid, e.g., 1 => 1D grid, 2 => 2D grid, 3=> 3D grid, etc.
#define CHANNELS 1
typedef float grid_precision;   // the type of values in the grid, e.g., float, double, int, etc.
typedef float func_precision;   // the type of values taken by the error function, e.g., float, double, int, etc.
typedef uint8_t pixel_precision; // the type of values in the image, e.g., float, double, int, etc.

#define DEPTH 1

// typedef func_byvalue_t<func_precision, grid_precision, grid_dimension,
//         CudaImage<pixel_precision, CHANNELS>, CudaImage<pixel_precision, CHANNELS> > image_err_func_byvalue;

// create device function pointer for by-value kernel function here
// __device__ image_err_func_byvalue dev_func_byvalue_ptr = averageAbsoluteDifference<func_precision, grid_precision, grid_dimension, CHANNELS, pixel_precision>;
//__device__ image_err_func_byvalue dev_func_byvalue_ptr = sumOfAbsoluteDifferences<func_precision, grid_precision, grid_dimension, pixel_precision>;

// SQD/NCC/MI
typedef func_byvalue_t<func_precision, grid_precision, grid_dimension,
        CudaImage<pixel_precision, CHANNELS>, CudaImage<pixel_precision, CHANNELS> > image_err_func_byvalue;
__device__ image_err_func_byvalue dev_func_byvalue_ptr = calcMIAlt<func_precision, grid_precision,
        grid_dimension, CHANNELS, pixel_precision>;

#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

void cxxopts_integration(cxxopts::Options &options) {

    options.add_options()
            ("i_ref", "Reference Image (image in the reference coordinate frame)", cxxopts::value<std::string>())
            ("i_mov", "Moved Image (image in the measured coordinate frame)", cxxopts::value<std::string>())
            ("d,debug", "Enable debugging", cxxopts::value<bool>()->default_value("false"))
            ("o,output", "Output file <output_image.png>",
             cxxopts::value<std::string>()->default_value("output_image.png"))
            ("f,fusedoutput", "Fused output file <output_image_fused.png>",
             cxxopts::value<std::string>()->default_value("output_image_fused.png"))
            ("h,help", "Print usage");
}

void printMatrix(double **matrix, int ROWS, int COLUMNS) {

    for (int r = 0; r < ROWS; r++) {
        for (int c = 0; c < COLUMNS; c++) {
            std::cout << matrix[r][c] << " ";
        }
        std::cout << std::endl;
    }
}

/*
// Generic functor
template<typename _Scalar, int NX = Eigen::Dynamic, int NY = Eigen::Dynamic>
struct Functor
{
    typedef _Scalar Scalar;
    enum {
        InputsAtCompileTime = NX,
        ValuesAtCompileTime = NY
    };
    typedef Eigen::Matrix<Scalar,InputsAtCompileTime,1> InputType;
    typedef Eigen::Matrix<Scalar,ValuesAtCompileTime,1> ValueType;
    typedef Eigen::Matrix<Scalar,ValuesAtCompileTime,InputsAtCompileTime> JacobianType;

    int m_inputs, m_values;

    Functor() : m_inputs(InputsAtCompileTime), m_values(ValuesAtCompileTime) {}
    Functor(int inputs, int values) : m_inputs(inputs), m_values(values) {}

    int inputs() const { return m_inputs; }
    int values() const { return m_values; }

};
*/
CudaImage<uint8_t, CHANNELS> *image_fix_test;
CudaImage<uint8_t, CHANNELS> *image_mov_test;
/*
struct my_functor : Functor<float>
{
    my_functor(void): Functor<float>(grid_dimension,grid_dimension) {}
    int operator()(const Eigen::VectorXf &x, Eigen::VectorXf &fvec) const
    {
        float minParams[grid_dimension] = {0};
        for (int i = 0; i < grid_dimension; i++)
            minParams[i] = x(i);
        nv_ext::Vec<float, grid_dimension> minParamsVec(minParams);

        fvec(0) = sqrt(calcNCCAlt<func_precision, grid_precision, grid_dimension, CHANNELS, pixel_precision>(minParamsVec, *image_fix_test, *image_mov_test));
        for (int i = 1; i < grid_dimension; i++)
            fvec(i) = 0;
        return 0;
    }
};
*/

// test grid search
// classes typically store images in column major format so the images
// stored are the transpose of that shown in initialization below
pixel_precision imageA_data[6 * 6] = {0, 0, 0, 0, 0, 0,
                                      0, 0, 0, 0, 0, 0,
                                      0, 0, 255, 255, 0, 0,
                                      0, 0, 255, 255, 0, 0,
                                      0, 0, 0, 0, 0, 0,
                                      0, 0, 0, 0, 0, 0};

pixel_precision imageB_data[6 * 6] = {0, 0, 0, 0, 0, 0,
                                      0, 0, 0, 0, 0, 0,
                                      0, 0, 0, 0, 0, 0,
                                      0, 0, 0, 0, 0, 0,
                                      0, 0, 0, 0, 1, 1,
                                      0, 0, 0, 0, 1, 1};

pixel_precision imageC_data[6 * 6] = {1, 1, 0, 0, 0, 0,
                                      1, 1, 0, 0, 0, 0,
                                      0, 0, 0, 0, 0, 0,
                                      0, 0, 0, 0, 0, 0,
                                      0, 0, 0, 0, 0, 0,
                                      0, 0, 0, 0, 0, 0
};

float delta_filter_data[1]={1.0f};

#define F_D5x5 25.0f
float avg_filter_5x5_data[5 * 5] = {1.0f / F_D5x5, 1.0f / F_D5x5, 1.0f / F_D5x5, 1.0f / F_D5x5, 1.0f / F_D5x5,
                                    1.0f / F_D5x5, 1.0f / F_D5x5, 1.0f / F_D5x5, 1.0f / F_D5x5, 1.0f / F_D5x5,
                                    1.0f / F_D5x5, 1.0f / F_D5x5, 1.0f / F_D5x5, 1.0f / F_D5x5, 1.0f / F_D5x5,
                                    1.0f / F_D5x5, 1.0f / F_D5x5, 1.0f / F_D5x5, 1.0f / F_D5x5, 1.0f / F_D5x5,
                                    1.0f / F_D5x5, 1.0f / F_D5x5, 1.0f / F_D5x5, 1.0f / F_D5x5, 1.0f / F_D5x5,
};

float central_diff_5[5] = {
        1.0f / 12.0f, -2.0f / 3.0f, 0.0f , 2.0f / 3.0f, -1.0f / 12.0f
};

#define F_D10x10 100.0f
float avg_filter_10x10_data[10 * 10] = {1.0f / F_D10x10, 1.0f / F_D10x10, 1.0f / F_D10x10, 1.0f / F_D10x10, 1.0f / F_D10x10,
                                        1.0f / F_D10x10, 1.0f / F_D10x10, 1.0f / F_D10x10, 1.0f / F_D10x10, 1.0f / F_D10x10,
                                        1.0f / F_D10x10, 1.0f / F_D10x10, 1.0f / F_D10x10, 1.0f / F_D10x10, 1.0f / F_D10x10,
                                        1.0f / F_D10x10, 1.0f / F_D10x10, 1.0f / F_D10x10, 1.0f / F_D10x10, 1.0f / F_D10x10,
                                        1.0f / F_D10x10, 1.0f / F_D10x10, 1.0f / F_D10x10, 1.0f / F_D10x10, 1.0f / F_D10x10,
                                        1.0f / F_D10x10, 1.0f / F_D10x10, 1.0f / F_D10x10, 1.0f / F_D10x10, 1.0f / F_D10x10,
                                        1.0f / F_D10x10, 1.0f / F_D10x10, 1.0f / F_D10x10, 1.0f / F_D10x10, 1.0f / F_D10x10,
                                        1.0f / F_D10x10, 1.0f / F_D10x10, 1.0f / F_D10x10, 1.0f / F_D10x10, 1.0f / F_D10x10,
                                        1.0f / F_D10x10, 1.0f / F_D10x10, 1.0f / F_D10x10, 1.0f / F_D10x10, 1.0f / F_D10x10,
                                        1.0f / F_D10x10, 1.0f / F_D10x10, 1.0f / F_D10x10, 1.0f / F_D10x10, 1.0f / F_D10x10,
                                        1.0f / F_D10x10, 1.0f / F_D10x10, 1.0f / F_D10x10, 1.0f / F_D10x10, 1.0f / F_D10x10,
                                        1.0f / F_D10x10, 1.0f / F_D10x10, 1.0f / F_D10x10, 1.0f / F_D10x10, 1.0f / F_D10x10,
                                        1.0f / F_D10x10, 1.0f / F_D10x10, 1.0f / F_D10x10, 1.0f / F_D10x10, 1.0f / F_D10x10,
                                        1.0f / F_D10x10, 1.0f / F_D10x10, 1.0f / F_D10x10, 1.0f / F_D10x10, 1.0f / F_D10x10,
                                        1.0f / F_D10x10, 1.0f / F_D10x10, 1.0f / F_D10x10, 1.0f / F_D10x10, 1.0f / F_D10x10,
                                        1.0f / F_D10x10, 1.0f / F_D10x10, 1.0f / F_D10x10, 1.0f / F_D10x10, 1.0f / F_D10x10,
                                        1.0f / F_D10x10, 1.0f / F_D10x10, 1.0f / F_D10x10, 1.0f / F_D10x10, 1.0f / F_D10x10,
                                        1.0f / F_D10x10, 1.0f / F_D10x10, 1.0f / F_D10x10, 1.0f / F_D10x10, 1.0f / F_D10x10,
                                        1.0f / F_D10x10, 1.0f / F_D10x10, 1.0f / F_D10x10, 1.0f / F_D10x10, 1.0f / F_D10x10,
                                        1.0f / F_D10x10, 1.0f / F_D10x10, 1.0f / F_D10x10, 1.0f / F_D10x10, 1.0f / F_D10x10
};

void display_data(float *a, int ax, int ay) {
    for (int y = 0; y < ay; y++) {
        for (int x = 0; x < ax; x++) {
            std::cout << a[x + y * ax] << ", ";
        }
        std::cout << std::endl;
    }
}

void conv2_data(float *a, int ax, int ay, float *h, int hx, int hy, float *c) {
    int ax0, ay0, hx0, hy0;
    for (ax0 = 0; ax0 < ax; ax0++) {
        for (ay0 = 0; ay0 < ay; ay0++) {
            for (hx0 = 0; hx0 < hx; hx0++) {
                for (hy0 = 0; hy0 < hy; hy0++) {
                    if (ax0 - hx0 >= 0 && ax0 - hx0 < ax && ay0 - hy0 >= 0 && ay0 - hy0 < ay) {
                        c[ax0 + ay0 * ax] += h[hx0 + hy0 * hx] * a[(ax0 - hx0) + (ay0 - hy0) * ax];
                    }
                }
            }
        }
    }
}

int main(int argc, char **argv) {

    // Argument parsing
    cxxopts::Options options("cuda_gridsearch", "UNC Charlotte Machine Vision Lab CUDA-accelerated grid search code.");
    cxxopts_integration(options);
    auto result = options.parse(argc, argv);
    std::string img_fixed_filename, img_moved_filename, img_out_filename, img_fused_filename;
    if (result.count("i_ref")) {
        img_fixed_filename = result["i_ref"].as<std::string>();
    } else {
        std::cerr << "No input reference image filename was provided. Exiting.." << std::endl;
        std::cout << options.help() << std::endl;
        return EXIT_FAILURE;
    }
    if (result.count("i_mov")) {
        img_moved_filename = result["i_mov"].as<std::string>();
    } else {
        std::cerr << "No input moving image filename was provided. Exiting.." << std::endl;
        std::cout << options.help() << std::endl;
        return EXIT_FAILURE;
    }
    img_out_filename = result["output"].as<std::string>();
    img_fused_filename = result["fusedoutput"].as<std::string>();
    std::cerr << "Output image filename is " << img_out_filename << "." << std::endl;
    std::cerr << "Fused Output image filename is " << img_fused_filename << "." << std::endl;
    if (result.count("help")) {
        std::cout << options.help() << std::endl;
        return EXIT_SUCCESS;
    }

    std::ofstream outfile;
    outfile.open("imageMatcherResults.txt", std::ios_base::app);
    outfile << "input files," << img_fixed_filename.c_str() << "," << img_moved_filename.c_str() << ",";
    outfile << "output fused image," << img_fused_filename << ",";
    outfile << "DEPTH," << DEPTH << ",";

    /* set GPU grid & block configuration */
    image_err_func_byvalue host_func_byval_ptr;
    int cuda_device = 0;
    hipDeviceProp_t deviceProp;

    cuda_device = findCudaDevice(0, nullptr);
    checkCudaErrors(hipGetDevice(&cuda_device));
    checkCudaErrors(hipGetDeviceProperties(&deviceProp, cuda_device));

    // Load input images from disk
    int xf, yf, nf;
    uint8_t *dataf = stbi_load(img_fixed_filename.c_str(), &xf, &yf, &nf, CHANNELS);
    if (dataf == NULL) {
        std::cerr << "Reference image " + img_fixed_filename + " failed to load!" << std::endl;
        return EXIT_FAILURE;
    }
    int xm, ym, nm;
    uint8_t *datam = stbi_load(img_moved_filename.c_str(), &xm, &ym, &nm, CHANNELS);
    if (datam == NULL) {
        std::cerr << "Moving image " + img_moved_filename + " failed to load!" << std::endl;
        return EXIT_FAILURE;
    }

    float MAX_SIZE_DISCREPANCY = 1.5;
    int x_new, y_new;
    uint8_t *data_new;
    float scale_factor_x = 1;
    float scale_factor_y = 1;
    if (xf * yf > MAX_SIZE_DISCREPANCY * xm * ym) { // resize fixed image
        x_new = xm;
        y_new = ym;
        scale_factor_x = (float)xm / (float)xf;
        scale_factor_y = (float)ym / (float)yf;

        std::cerr << "Rescaling " + img_fixed_filename + " from " << "(" << xf << "," << yf << ")" << " to "
                  << "(" << x_new << "," << y_new << ")" << std::endl;
        data_new = (uint8_t *) malloc(x_new * y_new * CHANNELS);
        stbir_resize_uint8(dataf, xf, yf, 0, data_new, x_new, y_new, 0, CHANNELS);
        if (data_new == NULL) {
            std::cerr << "Image resize " + img_fixed_filename + " failed!" << std::endl;
            return EXIT_FAILURE;
        }
        xf = x_new;
        yf = y_new;
        stbi_image_free(dataf);
        dataf = data_new;
        stbi_write_png("resized_image.png", x_new, y_new, CHANNELS, data_new,
                       x_new * sizeof(uint8_t) * CHANNELS);
    } else if (xm * ym > MAX_SIZE_DISCREPANCY * xf * yf) {
        x_new = xf;
        y_new = yf;
        std::cerr << "Rescaling " + img_moved_filename + " from " << "(" << xf << "," << yf << ")" << " to "
                  << "(" << x_new << "," << y_new << ")" << std::endl;
        data_new = (uint8_t *) malloc(x_new * y_new * CHANNELS);
        stbir_resize_uint8(datam, xm, ym, 0, data_new, x_new, y_new, 0, CHANNELS);
        if (data_new == NULL) {
            std::cerr << "Image resize " + img_moved_filename + " failed!" << std::endl;
            return EXIT_FAILURE;
        }
        xm = x_new;
        ym = y_new;
        stbi_image_free(datam);
        datam = data_new;
        stbi_write_png("resized_image.png", x_new, y_new, CHANNELS, data_new,
                       x_new * sizeof(uint8_t) * CHANNELS);
    }
    // number of components must be equal on construction
    // assert(nf == CHANNELS && nm == CHANNELS); // Does not work if using gray scale, nf/nm are based on original channels
    outfile << "Scale Factor," << scale_factor_x << "," << scale_factor_y << ",";
    CudaImage<uint8_t, CHANNELS> image_fix(yf, xf);
    CudaImage<uint8_t, CHANNELS> image_mov(ym, xm);

    checkCudaErrors(hipMalloc(&image_fix.data(), image_fix.bytesSize()));
    checkCudaErrors(hipMemcpy(image_fix.data(), dataf, image_fix.bytesSize(), hipMemcpyHostToDevice));

    checkCudaErrors(hipMalloc(&image_mov.data(), image_mov.bytesSize()));
    checkCudaErrors(hipMemcpy(image_mov.data(), datam, image_mov.bytesSize(), hipMemcpyHostToDevice));

    CudaImage<float> delta_filter(1, 1);
    checkCudaErrors(hipMalloc(&delta_filter.data(), delta_filter.bytesSize()));
    delta_filter.setValuesFromVector(std::vector<float>(delta_filter_data, delta_filter_data + 1 * 1));

    CudaImage<float> avg_filter_5x5(5, 5);
    checkCudaErrors(hipMalloc(&avg_filter_5x5.data(), avg_filter_5x5.bytesSize()));
    avg_filter_5x5.setValuesFromVector(std::vector<float>(avg_filter_5x5_data, avg_filter_5x5_data + 5 * 5));
    // avg_filter_5x5.display("avg_filter_5x5");

    CudaImage<float> avg_filter_10x10(10, 10);
    checkCudaErrors(hipMalloc(&avg_filter_10x10.data(), avg_filter_10x10.bytesSize()));
    avg_filter_10x10.setValuesFromVector(std::vector<float>(avg_filter_10x10_data, avg_filter_10x10_data + 10 * 10));

    float zeros_5x5[5*5] =  { 0 };
    float sobel_5x5[5*5] =  { 0 };
    // set element [2,0] to 1
    zeros_5x5[2*5+0] = 1.0f;
    display_data(zeros_5x5, 5, 5);
    conv2_data(zeros_5x5, 5, 5, central_diff_5, 5, 1, sobel_5x5);
    display_data(sobel_5x5, 5, 5);
    zeros_5x5[2*5+0] = 0.0f;
    zeros_5x5[0*5+2] = 1.0f;
    display_data(zeros_5x5, 5, 5);
    conv2_data(zeros_5x5, 5, 5, central_diff_5, 1, 5, sobel_5x5);
    zeros_5x5[0*5+2] = 0.0f;
    display_data(sobel_5x5, 5, 5);
    for (int idx=0; idx < 5; idx++) {
        sobel_5x5[idx + idx * 5] += central_diff_5[idx];
    }
    display_data(sobel_5x5, 5, 5);
    CudaImage<float> sobel_filter_5x5(10, 10);
    checkCudaErrors(hipMalloc(&sobel_filter_5x5.data(), sobel_filter_5x5.bytesSize()));
    sobel_filter_5x5.setValuesFromVector(std::vector<float>(sobel_5x5, sobel_5x5 + 5 * 5));

    CudaImage<uint8_t, CHANNELS> image_fix_filtered(yf, xf);
    checkCudaErrors(hipMalloc(&image_fix_filtered.data(), image_fix_filtered.bytesSize()));
    CHANNEL_ACTION actions[CHANNELS] {FILTER};

    // image_fix.filter(delta_filter, image_fix_filtered, actions);
    // image_fix.filter(avg_filter_5x5, image_fix_filtered, actions);
    image_fix.filter(avg_filter_10x10, image_fix_filtered, actions);
    // image_fix.filter(sobel_filter_5x5, image_fix_filtered, actions);

    if (DEBUG) {
        // Write an output image to disk
        float identityH_data[] = {1.0f, 0.0f, 0.0f,
                                  0.0f, 1.0f, 0.0f,
                                  0.0f, 0.0f};
        nv_ext::Vec<float, 8> identityH(identityH_data);
        writeTransformedImageToDisk<uint8_t, CHANNELS>(image_fix, yf, xf, identityH, "image_fixed.png");
        writeTransformedImageToDisk<uint8_t, CHANNELS>(image_fix_filtered, yf, xf, identityH,
                                                       "image_fixed_filtered.png");
    }

    CudaImage<uint8_t, CHANNELS> image_fix2(yf, xf);
    CudaImage<uint8_t, CHANNELS> image_mov2(ym, xm);

    image_fix2.data() = dataf;
    image_mov2.data() = datam;

    image_fix_test = &image_fix2;
    image_mov_test = &image_mov2;

    checkCudaErrors(hipDeviceSetLimit(hipLimitMallocHeapSize, 1 << 30));

    //    linear interpolation in homography / affine matrix space
    //    https://math.stackexchange.com/questions/612006/decomposing-an-affine-transformation
    //    using the parameterization described by Stephane Laurent
    float theta = 5.0 * PI / 180.0; // range [0, 2*PI]
    float scaleX = 1.5;  // // range [1, 2]
    float scaleY = 1.5;  // // range [1, 2]
    float shearXY = 0.2; // range [-0.2, 0.2]
    float translateX = -10; // range [-image.width()/2, image.width()/2]
    float translateY = -30; // range [-image.height()/2, image.height()/2]
    float keystoneX = 0.0; // range [-0.1, 0.1]
    float keystoneY = 0.0; // range [-0.1, 0.1]
    // Transform does scale, shear, rotate, then translate and finally perspective project per the website
    // https://math.stackexchange.com/questions/612006/decomposing-an-affine-transformation
    //float initialH[] = {1.0, 0.0, 0.0, 0.0, 1.0, 0.0, 0.0, 0.0};

    float initialH[] = {scaleX * cos(theta), scaleY * shearXY * cos(theta) - scaleY * sin(theta), scaleX * translateX,
                        scaleX * sin(theta), scaleY * shearXY * sin(theta) + scaleY * cos(theta), scaleY * translateY,
                        keystoneX, keystoneY};

    // Example MI
    scaleX = 1;
    scaleY = 1;
    float MAX_NONOVERLAPPING_PCT = 0.5f;
    std::vector<grid_precision> start_point = {(grid_precision) -PI / 40, // theta
                                               (grid_precision) 1.5 * scale_factor_x, // scaleX
                                               (grid_precision) 1.5 * scale_factor_y, // scaleY
                                               (grid_precision) -0.4,  // shearXY
                                               (grid_precision) -xm * MAX_NONOVERLAPPING_PCT,  // translateX
                                               (grid_precision) -ym * MAX_NONOVERLAPPING_PCT,  // translateY
                                               (grid_precision) 0, // keystoneX
                                               (grid_precision) 0  // keystoneY
    };

    std::vector<grid_precision> num_samples = {(grid_precision) 32,
                                               (grid_precision) 16,
                                               (grid_precision) 16,
                                               (grid_precision) 5,
                                               (grid_precision) (xf + 1) / (20 * scale_factor_x),
                                               (grid_precision) (yf + 1) / (20 * scale_factor_y),
                                               (grid_precision) 1,
                                               (grid_precision) 1
    };

    std::vector<grid_precision> end_point = {static_cast<float>((grid_precision) 2 * PI - PI / num_samples[0]),
                                             (grid_precision) 5 * scale_factor_x,
                                             (grid_precision) 5 * scale_factor_y,
                                             (grid_precision) 0.2,
                                             (grid_precision) xf - xm * MAX_NONOVERLAPPING_PCT,
                                             (grid_precision) yf - ym * MAX_NONOVERLAPPING_PCT,
                                             (grid_precision) 0,
                                             (grid_precision) 0
    };

    outfile << "start_point,";
    for (int i = 0; i < grid_dimension; i++) outfile << start_point[i] << ",";
    outfile << "end_point,";
    for (int i = 0; i < grid_dimension; i++) outfile << end_point[i] << ",";
    outfile << "num_samples,";
    for (int i = 0; i < grid_dimension; i++) outfile << num_samples[i] << ",";
    float minParams[grid_dimension] = {0};

    std::chrono::time_point<std::chrono::system_clock> start, end;
    start = std::chrono::system_clock::now();
    for (int iii = 0; iii < DEPTH; iii++) {
        CudaGrid<grid_precision, grid_dimension> translation_xy_grid;
        checkCudaErrors(hipMalloc(&translation_xy_grid.data(), translation_xy_grid.bytesSize()));

        translation_xy_grid.setStartPoint(start_point);
        translation_xy_grid.setEndPoint(end_point);
        translation_xy_grid.setNumSamples(num_samples);
        translation_xy_grid.display("translation_xy_grid");

        grid_precision axis_sample_counts[grid_dimension];
        translation_xy_grid.getAxisSampleCounts(axis_sample_counts);

        CudaTensor<func_precision, grid_dimension> func_values(axis_sample_counts);
        checkCudaErrors(hipMalloc(&func_values.data(), func_values.bytesSize()));
        //func_values.fill(0);

        // first template argument is the error function return type
        // second template argument is the grid point value type
        CudaGridSearcher<func_precision, grid_precision, grid_dimension> translation_xy_gridsearcher(
                translation_xy_grid,
                func_values);

        // Mutual Information
        // Copy device function pointer for the function having by-value parameters to host side
        hipMemcpyFromSymbol(&host_func_byval_ptr, HIP_SYMBOL(dev_func_byvalue_ptr),
                             sizeof(image_err_func_byvalue));

        //translation_xy_gridsearcher.search(host_func_byval_ptr, m1, m2);
        translation_xy_gridsearcher.search_by_value(host_func_byval_ptr, image_mov, image_fix);
        // translation_xy_gridsearcher.search_by_value_stream(host_func_byval_ptr, 10000, 1, image_mov, image_fix);
        // translation_xy_gridsearcher.search_by_value_stream(host_func_byval_ptr, 10000, image_fix.height(), image_mov, image_fix);

        //    func_values.display();

        func_precision min_value;
        int32_t min_value_index1d;
        func_values.find_extrema(min_value, min_value_index1d);

        grid_precision min_grid_point[grid_dimension];
        translation_xy_grid.getGridPoint(min_grid_point, min_value_index1d);
        std::cout << "Minimum found at point p = { ";
        for (int d = 0; d < grid_dimension; d++) {
            std::cout << min_grid_point[d] << ((d < grid_dimension - 1) ? ", " : " ");
            minParams[d] = min_grid_point[d];

            start_point[d] = min_grid_point[d] - (end_point[d] - start_point[d]) / 4;
            end_point[d] = min_grid_point[d] + (end_point[d] - start_point[d]) / 4;
        }
        std::cout << "}" << std::endl;

        checkCudaErrors(hipFree(translation_xy_grid.data()));
        checkCudaErrors(hipFree(func_values.data()));
    }
    end = std::chrono::system_clock::now();
    std::chrono::duration<double> elapsed_seconds = end - start;
    printf("Total Time Taken: %f\n", elapsed_seconds.count());

    outfile << "Total Time," << elapsed_seconds.count() << ",";

    outfile << "MinParams,";
    for (int i = 0; i < grid_dimension; i++) outfile << minParams[i] << ",";
    // Non-linear optimizer
    // Eigen::VectorXf x(grid_dimension);
    // for(int i = 0; i < grid_dimension; i++)
    //     x(i) = minParams[i];
    // std::cout << "x: " << x << std::endl;

    // my_functor functor;
    // Eigen::NumericalDiff<my_functor> numDiff(functor);
    // Eigen::LevenbergMarquardt<Eigen::NumericalDiff<my_functor>,float> lm(numDiff);
    // lm.parameters.maxfev = 2000;
    //lm.parameters.xtol = 1.0e-10;

    // int ret = lm.minimize(x);
    // std::cout << "Iterations: " << lm.iter << ", Return code: " << ret << std::endl;

    // std::cout << "x that minimizes the function: " << x << std::endl;

    // Convert min Values to H
    // for(int i = 0; i < grid_dimension; i++)
    //     minParams[i] = x(i);

    float h11 = 0, h12 = 0, h13 = 0, h21 = 0, h22 = 0, h23 = 0, h31 = 0, h32 = 0;
    float cx = (float) xm / 2, cy = (float) ym / 2;
    nv_ext::Vec<float, grid_dimension> minParamsVec(minParams);
    //std::cout << "Min Value check: " << calcMIAlt<func_precision, grid_precision, grid_dimension, CHANNELS, pixel_precision>(minParamsVec, *image_fix_test, *image_mov_test) << std::endl;
    parametersToHomography<grid_precision, grid_dimension>(minParamsVec, cx, cy,
                                                           h11, h12, h13,
                                                           h21, h22, h23,
                                                           h31, h32);
    float minH[] = {h11, h12, h13,
                    h21, h22, h23,
                    h31, h32};
    outfile << "Homography,";
    for (int i = 0; i < grid_dimension; i++) outfile << minH[i] << ",";
    outfile << "\n";
//    nv_ext::Vec<float, 8> H(initialH);
    nv_ext::Vec<float, 8> H(minH);

    // Write an output image to disk
    writeTransformedImageToDisk<uint8_t, CHANNELS>(image_mov, yf, xf, H, img_out_filename);

    // Write aligned and fused output image to disk
    writeAlignedAndFusedImageToDisk<uint8_t, CHANNELS>(image_fix, image_mov, H, H, img_fused_filename);

    // Clean memory
    checkCudaErrors(hipFree(image_fix.data()));
    checkCudaErrors(hipFree(image_mov.data()));


    stbi_image_free(dataf);
    stbi_image_free(datam);
    return EXIT_SUCCESS;
}
